#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <cassert>
#include <iostream>

#include "../math/math.hpp" // det3
#include "test.hpp"
#include "../include/globals.hpp" // get_tid, get_stride


__global__
void calc_det3(float const* matrices, unsigned int const num_matrices, float* dets) {
	for (auto tid = get_tid(); tid < num_matrices; tid += get_stride()) {
		float vals[9] = { 0 };

		for (int i = 0; i < 9; ++i) {
			// for the sake of maintenance,
			// i = row index
			// num_matrices = total number of columns
			// tid = column index
			vals[i] = matrices[i * num_matrices + tid];
		}

		dets[tid] = det3(vals);
	}
}

__global__
void calc_det4(float const* matrices, unsigned int const num_matrices, float* dets) {
	for (auto tid = get_tid(); tid < num_matrices; tid += get_stride()) {
		float vals[16] = { 0 };

		for (int i = 0; i < 16; ++i) {
			vals[i] = matrices[i * num_matrices + tid];
		}

		dets[tid] = det4(vals);
	}
}

void determinant_tests(void) {
	// we use some number of matrices...
	unsigned int const num_matrices = (1 << 8);

	/**
	 * We should be able to take the determinant of a 3x3 matrix
	 */
	{
		// we use 9 randomly generated values
		// the expected determinant in this case is: -11263739881452.000
		float const matrix_values[9] = { 87432, 84228, 93841, 65001, 51520, 65406, 23748, 48720, 47247 };

		thrust::device_vector<float> device_matrices{9 * num_matrices};
		thrust::host_vector<float> host_floats{9 * num_matrices};

		thrust::device_vector<float> device_dets{num_matrices};
		thrust::host_vector<float> host_dets{num_matrices};

		// for the sake of reliable testing, each matrix is the same
		for (int i = 0; i < 9; ++i) { // 9 rows
			for (int j = 0; j < num_matrices; ++j) {
				host_floats[i * num_matrices + j] = matrix_values[i];
			}
		}

		device_matrices = host_floats;

		calc_det3<<<bpg, tpb>>>(
				device_matrices.data().get(),
				num_matrices,
				device_dets.data().get());
		hipDeviceSynchronize();

		host_dets = device_dets;

		for (auto &v : host_dets) {
			assert(static_cast<long int>(v) == -11263705874432);
		}
	}

	/**
	 * We should be able to take the determinant of a 4x4
	 */
	{
		// expect det is -5813662.000
		float const matrix_values[16] = {
				63, 90, 13, 83,
				73, 57, 78, 91,
				87, 15, 26, 23,
				66, 97, 22, 80
		};

		thrust::device_vector<float> device_matrices{16 * num_matrices};
		thrust::host_vector<float> host_floats{16 * num_matrices};

		thrust::device_vector<float> device_dets{num_matrices};
		thrust::host_vector<float> host_dets{num_matrices};

		// for the sake of reliable testing, each matrix is the same
		for (int i = 0; i < 16; ++i) { // 16 rows
			for (int j = 0; j < num_matrices; ++j) {
				host_floats[i * num_matrices + j] = matrix_values[i];
			}
		}

		device_matrices = host_floats;

		calc_det4<<<bpg, tpb>>>(
				device_matrices.data().get(),
				num_matrices,
				device_dets.data().get());
		hipDeviceSynchronize();

		host_dets = device_dets;

		for (auto &v : host_dets) {
			assert(static_cast<long int>(v) == -5813662);
		}
	}
}

