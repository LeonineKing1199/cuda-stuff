#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/fill.h>
#include <thrust/tuple.h>
#include <thrust/sort.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/unique.h>
#include <thrust/for_each.h>
#include <thrust/pair.h>
#include <thrust/distance.h>
#include <thrust/transform.h>

#include "../include/globals.hpp"
#include "../include/lib/nominate.hpp"

using thrust::device_vector;
using thrust::fill;
using thrust::tuple;
using thrust::get;
using thrust::make_zip_iterator;
using thrust::make_tuple;
using thrust::sort;
using thrust::unique_by_key_copy;
using thrust::for_each;
using thrust::pair;
using thrust::distance;
using thrust::transform;

/**
  * This function is used to determine which points will
  * be used in this round of insertion.
  * nm is an array aligned to the number of points that
  * are available
  * pa, ta, la are the association tuple
*/

auto nominate(
  int const assoc_size,
  device_vector<int>& pa,
  device_vector<int>& ta,
  device_vector<int>& la,
  device_vector<int>& nm) -> void
{
  // the first thing we want to do is sort everything
  // by ta
  auto zip_begin =
    make_zip_iterator(
      make_tuple(
        pa.begin(),
        ta.begin(),
        la.begin()));
        
  sort(
    zip_begin, zip_begin + assoc_size,
    [] __device__ (
      tuple<int, int, int> const& a,
      tuple<int, int, int> const& b) -> bool
    {
      int const a_ta_id{get<1>(a)};
      int const a_pa_id{get<0>(a)};
      
      int const b_ta_id{get<1>(b)};
      int const b_pa_id{get<0>(b)};
      
      return a_ta_id == b_ta_id ? a_pa_id < b_pa_id : a_ta_id < b_ta_id;
    });
 
  
  // we then want to allocate copies of our
  // association arrays to write our stream
  // compaction to
  device_vector<int> pa_cpy{assoc_size, -1};
  device_vector<int> ta_cpy{assoc_size, -1};
    
  // remove tuple elements, using ta as the
  // unique key
  auto last_pair = unique_by_key_copy(
    ta.begin(), ta.begin() + assoc_size,
    pa.begin(),
    ta_cpy.begin(),
    pa_cpy.begin());

  // unique_by_key_copy returns a pair of iterators (keys_last, values_last)
  int const assoc_cpy_size{static_cast<int>(distance(ta_cpy.begin(), last_pair.first))};
  
  fill(nm.begin(), nm.end(), 0);
  device_vector<int> nm_cpy{nm};
  
  int* nm_data = nm.data().get();
  int* nm_cpy_data = nm_cpy.data().get();
  
  // this is how we count the number of occurrences for a particular
  // point index
  // if the copy doesn't match up with the original count array, that
  // means that the point had some non-unique tetrahedra associated
  // with it and as such is not up for nomination
  for_each(
    pa.begin(), pa.begin() + assoc_size,
    [=] __device__ (int const pa_id) -> void
    {
      atomicAdd(nm_data + pa_id, 1);
    });
    
  for_each(
    pa_cpy.begin(), pa_cpy.begin() + assoc_cpy_size,
    [=] __device__ (int const pa_id) -> void
    {
      atomicAdd(nm_cpy_data + pa_id, 1);
    });
    
  // we perform a simple transformation over both ranges and
  // check for equality.
  // if the point occurred the same amount of times then all
  // of its  tetrahedra were unique and is able to be nominated
  transform(
    nm.begin(), nm.end(),
    nm_cpy.begin(),
    nm.begin(),
    [] __device__ (int const a, int const b) -> int
    {
      return (a != 0) && (a - b == 0);
    });//*/
}
