#include "hip/hip_runtime.h"
#include "../include/lib/nominate.hpp"

__global__
void nominate(
  int const assoc_size,
  int const* __restrict__ ta,
  int const* __restrict__ pa,
  int* nm_ta,
  int* nm)
{
  for (auto tid = get_tid(); tid < assoc_size; tid += grid_stride()) {
    int* address = nm_ta + ta[tid];
    int compare = -1;
    int val = tid;
    
    // this thread was the first one to find this tetrahedron
    if (atomicCAS(address, compare, val) == -1) {
      // we then want to nominate this point
      // but because we initialize pa to being all true, if any
      // entry was previously 0, we know it was marked false by
      // another thread so we set it back to being false
      if (atomicOr(nm + pa[tid], 1) == 0) {
        atomicAnd(nm + pa[tid], 0);
      }
    } else {
      atomicAnd(nm + pa[tid], 0);
    }
  }
}