#include "hip/hip_runtime.h"
#include "globals.hpp"
#include "size_type.hpp"
#include "index_t.hpp"
#include "lib/nominate.hpp"

#include <thrust/device_vector.h>
#include <thrust/fill.h>
#include <thrust/tuple.h>
#include <thrust/sort.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/unique.h>
#include <thrust/for_each.h>
#include <thrust/pair.h>
#include <thrust/distance.h>
#include <thrust/transform.h>

using thrust::device_vector;
using thrust::fill;
using thrust::tuple;
using thrust::get;
using thrust::make_zip_iterator;
using thrust::make_tuple;
using thrust::sort;
using thrust::unique_by_key_copy;
using thrust::for_each;
using thrust::pair;
using thrust::distance;
using thrust::transform;

/**
  * This function is used to determine which points will
  * be used in this round of insertion.
  * nm is an array aligned to the number of points that
  * are available
  * pa, ta, la are the association tuple
*/

auto nominate(
  size_t const assoc_size,
  thrust::device_vector<index_t>& pa,
  thrust::device_vector<index_t>& ta,
  thrust::device_vector<index_t>& la,
  thrust::device_vector<unsigned>& nm) -> void
{
  // the first thing we want to do is sort everything
  // by ta
  auto zip_begin =
    make_zip_iterator(
      make_tuple(
        pa.begin(),
        ta.begin(),
        la.begin()));
        
  sort(
    zip_begin, zip_begin + assoc_size,
    [] __device__ (
      tuple<index_t, index_t, index_t> const& a,
      tuple<index_t, index_t, index_t> const& b) -> bool
    {
      index_t const a_ta_id{get<1>(a)};
      index_t const a_pa_id{get<0>(a)};
      
      index_t const b_ta_id{get<1>(b)};
      index_t const b_pa_id{get<0>(b)};
      
      return (a_ta_id == b_ta_id) ? (a_pa_id < b_pa_id) : (a_ta_id < b_ta_id);
    });
 
  
  // we then want to allocate copies of our
  // association arrays to write our stream
  // compaction to
  device_vector<index_t> pa_cpy{assoc_size};
  device_vector<index_t> ta_cpy{assoc_size};
    
  // remove tuple elements, using ta as the
  // unique key
  auto last_pair = unique_by_key_copy(
    ta.begin(), ta.begin() + assoc_size,
    pa.begin(),
    ta_cpy.begin(),
    pa_cpy.begin());

  // unique_by_key_copy returns a pair of iterators (keys_last, values_last)
  size_t const assoc_cpy_size{static_cast<size_t>(distance(ta_cpy.begin(), last_pair.first))};
  
  fill(nm.begin(), nm.end(), 0);
  device_vector<unsigned> nm_cpy{nm};
  
  unsigned* nm_data = nm.data().get();
  unsigned* nm_cpy_data = nm_cpy.data().get();
  
  // this is how we count the number of occurrences for a particular
  // point index
  // if the copy doesn't match up with the original count array, that
  // means that the point had some non-unique tetrahedra associated
  // with it and as such is not up for nomination
  for_each(
    pa.begin(), pa.begin() + assoc_size,
    [=] __device__ (index_t const pa_id) -> void
    {
      atomicAdd(nm_data + static_cast<unsigned long long>(pa_id), 1);
    });
    
  for_each(
    pa_cpy.begin(), pa_cpy.begin() + assoc_cpy_size,
    [=] __device__ (index_t const pa_id) -> void
    {
      atomicAdd(nm_cpy_data + static_cast<unsigned long long>(pa_id), 1);
    });
    
  // we perform a simple transformation over both ranges and
  // check for equality.
  // if the point occurred the same amount of times then all
  // of its  tetrahedra were unique and is able to be nominated
  transform(
    nm.begin(), nm.end(),
    nm_cpy.begin(),
    nm.begin(),
    [] __device__ (unsigned const a, unsigned const b) -> unsigned
    {
      return (a != 0) && (a - b == 0);
    });//*/
}
