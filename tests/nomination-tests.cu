#include <thrust/device_vector.h>

#include "test-suite.hpp"
#include "../include/lib/nominate.hpp"

auto nomination_tests(void) -> void
{
  std::cout << "Beginning nomination tests!" << std::endl;
  
  // We should be able to nominate points for a given
  // association configuration
  {
    int const num_pts = 4;
    int const num_tets = 8;
    
    thrust::device_vector<int> nm{ num_pts, 1 };
    
    thrust::device_vector<int> ta{ 10 };
    thrust::device_vector<int> pa{ 10 };
    
    ta[0] = 0;
    ta[1] = 1;
    ta[2] = 2;
    ta[3] = 3;
    ta[4] = 4;
    ta[5] = 5;
    ta[6] = 0;
    ta[7] = 2;
    ta[8] = 6;
    ta[9] = 7;
    
    pa[0] = 0;
    pa[1] = 0;
    pa[2] = 0;
    pa[3] = 0;
    pa[4] = 1;
    pa[5] = 1;
    pa[6] = 2;
    pa[7] = 3;
    pa[8] = 3;
    pa[9] = 3;
    
    thrust::device_vector<int> nm_ta{num_tets, 0};
    
    nominate<float><<<bpg, tpb>>>(
      10,
      ta.data().get(),
      pa.data().get(),
      nm_ta.data().get(),
      nm.data().get());
      
    hipDeviceSynchronize();
    
    assert(nm[1] == 1);
    
    if (nm[0] == 0) {
      assert(nm[2] == 1 && nm[3] == 1);
    } else {
      assert(nm[2] == 0 && nm[3] == 0);
    }
  }
  
  std::cout << "Tests Passed!\n" << std::endl;
}