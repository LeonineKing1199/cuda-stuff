#include "hip/hip_runtime.h"
#include "catch.hpp"
#include "globals.hpp"
#include "index_t.hpp"
#include "lib/nominate.hpp"
#include "math/rand-int-range.hpp"

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

using thrust::device_vector;
using thrust::host_vector;

TEST_CASE("The nomination function")
{
  /*size_type assoc_size = 11;
  size_type const ta_data[11] = { 0, 1, 2, 3, 2, 5, 6, 7, 8, 1, 8 };
  size_type const pa_data[11] = { 0, 0, 0, 0, 2, 2, 3, 3, 3, 4, 4 };
  
  host_vector<size_type> h_ta{ta_data, ta_data + 11};
  host_vector<size_type> h_pa{pa_data, pa_data + 11};
  
  device_vector<size_type> ta{h_ta};
  device_vector<size_type> pa{h_pa};
  device_vector<size_type> la{static_cast<usize_type>(assoc_size), -1};

  //*/
}

/*
__global__
void assert_unique(
  int const assoc_size,
  int const* __restrict__ pa,
  int const* __restrict__ nm,
  int const* __restrict__ ta,
  int* __restrict__ nm_ta)
{
  for (auto tid = get_tid(); tid < assoc_size; tid += grid_stride()) {
    if (nm[pa[tid]]) {
      assert(atomicCAS(nm_ta + ta[tid], -1, 1) == -1);
    }
  }
}

auto nomination_tests(void) -> void
{
  std::cout << "Beginning nomination tests!" << std::endl;
  
  {


    device_vector<int> nm{5, 0};

    nominate(assoc_size, pa, ta, la, nm);
    
    device_vector<int> nm_ta{9, -1};
    
    assert_unique<<<bpg, tpb>>>(
      assoc_size,
      pa.data().get(),
      nm.data().get(),
      ta.data().get(),
      nm_ta.data().get());
    
    hipDeviceSynchronize();
    
    /*for (unsigned i = 0; i < nm.size(); ++i) {
      std::cout << nm[i] << " ";
    }
    std::cout << "\n";//
  }
  
  {
    int assoc_size{5000};
    
    int const min{0};
    int const max{2500};
    
    device_vector<int> pa{rand_int_range(min, max, assoc_size, 0)};
    device_vector<int> ta{rand_int_range(min, max, assoc_size, assoc_size)};
    device_vector<int> la{assoc_size, -1};
    
    assert(pa.size() == static_cast<unsigned>(assoc_size));
    assert(ta.size() == static_cast<unsigned>(assoc_size));
    
    int const num_pts{max};
    device_vector<int> nm{num_pts, 0};
    
    nominate(assoc_size, pa, ta, la, nm);
    
    device_vector<int> nm_ta{max, -1};
    
    assert_unique<<<bpg, tpb>>>(
      assoc_size,
      pa.data().get(),
      nm.data().get(),
      ta.data().get(),
      nm_ta.data().get());
    
    hipDeviceSynchronize();
    
    /*for (unsigned i = 0; i < nm.size(); ++i) {
      std::cout << nm[i];
    }
    std::cout << "\n";//
  }
  
  std::cout << "Tests Passed!\n" << std::endl;
}*/