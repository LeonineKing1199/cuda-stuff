#include "hip/hip_runtime.h"
#include <thrust/transform.h>
#include <thrust/execution_policy.h>

#include "test-suite.hpp"
#include "../include/array.hpp"

__host__ __device__
auto array_tests_impl(void) -> void
{
  // we should  be able to construct an array
  {
    reg::array<float, 4> a{ 1.0f, 2.0f, 3.0f, 4.0f };
    reg::array<float, 4> b = a;
    
    assert((a == b));
    
    assert(a[0] == 1.0f);
    assert(a[1] == 2.0f);
    assert(a[2] == 3.0f);
    assert(a[3] == 4.0f);
    
    b[3] = 17.0f;
    
    assert((a != b));
  }
  
  // it should be transformable
  {
    reg::array<float, 4> a{ 1.0f, 2.0f, 3.0f, 4.0f };
    reg::array<float, 4> b{ 0 };

    thrust::transform(
      thrust::seq,
      a.begin(), a.end(),
      b.begin(),
      [](float const f) -> float
      {
        return f * f;
      });
    
    assert((b == reg::array<float, 4>{ 1.0f, 4.0f, 9.0f, 16.0f }));
  }
}

__global__
void test_kernel(void)
{
  array_tests_impl();
}

auto array_tests(void) -> void
{
  std::cout << "Beginning array tests!" << std::endl;
  
  array_tests_impl();

  // we should be able to do everything on the device as well
  {
    test_kernel<<<1, 256>>>();
    hipDeviceSynchronize();
  }
  
  std::cout << "Array tests passed!\n" << std::endl;
}