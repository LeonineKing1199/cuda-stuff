#include <thrust/reduce.h>
#include <thrust/extrema.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>

#include "regulus/array.hpp"
#include "regulus/algorithm/location.hpp"
#include "regulus/algorithm/nominate.hpp"
#include "regulus/utils/make_rand_range.hpp"

#include <catch.hpp>

TEST_CASE("Nominating points...")
{
  SECTION("should work as expected")
  {
    auto const assoc_size = std::size_t{11};

    using array_t = regulus::array<std::ptrdiff_t, assoc_size>;

    auto const ta_data = array_t{0, 1, 2, 3, 2, 5, 6, 7, 8, 1, 8};
    auto const pa_data = array_t{0, 0, 0, 0, 2, 2, 3, 3, 3, 4, 4};

    auto ta = thrust::device_vector<std::ptrdiff_t>{ta_data.begin(), ta_data.end()};
    auto pa = thrust::device_vector<std::ptrdiff_t>{pa_data.begin(), pa_data.end()};
    auto la = thrust::device_vector<regulus::loc_t>{assoc_size, regulus::outside_v};
    auto nm = thrust::device_vector<bool>{
      static_cast<std::size_t>(1 + *thrust::max_element(pa_data.begin(), pa_data.end())),
      false};

    regulus::nominate(assoc_size, pa, ta, la, nm);
    hipDeviceSynchronize();

    auto h_ta = thrust::host_vector<std::ptrdiff_t>{ta};
    auto h_pa = thrust::host_vector<std::ptrdiff_t>{pa};
    auto h_nm = thrust::host_vector<bool>{nm};

    auto nominated_cnt = thrust::host_vector<unsigned>{
      static_cast<std::size_t>(1 + *thrust::max_element(h_ta.begin(), h_ta.end())), 0};

    auto found_duplicate = false;
    auto num_nominated   = int{0};

    for (std::size_t i = 0; i < assoc_size; ++i) {
      if (h_nm[h_pa[i]]) {
        if (++nominated_cnt[h_ta[i]] > 1) {
          found_duplicate = true;
        }
        ++num_nominated;
      }
    }

    REQUIRE(found_duplicate == false);
    REQUIRE(num_nominated > 0);
    REQUIRE((
      thrust::reduce(
        nominated_cnt.begin(),
        nominated_cnt.end()) > 0));
  }
}