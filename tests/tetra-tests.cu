#include "hip/hip_runtime.h"
#include "test-suite.hpp"
#include "../include/math/tetra.hpp"

__host__ __device__
auto tetra_tests_impl(void) -> void
{
  using real = float;
  using point_t = point_t<real>;
  
  // We should be able to determine the orientation
  // of a tetrahedron correctly
  {    
    point_t const a{ 0.0, 0.0, 0.0 };
    point_t const b{ 9.0, 0.0, 0.0 };
    point_t const c{ 0.0, 9.0, 0.0 };
    point_t const d{ 0.0, 0.0, 9.0 };
    
    assert(orient<real>(a, b, c, d) == orientation::positive);
    
    point_t const e{ 3.0, 3.0, 0.0 };
    assert(orient<real>(a, b, c, e) == orientation::zero);
    
    point_t const f{ 3.0, 3.0, -3.0 };
    assert(orient<real>(a, b, c, f) == orientation::negative);
  }
  
  // insphere stuff should work as well
  {
    point_t const a{ 0.0, 0.0, 0.0 };
    point_t const b{ 9.0, 0.0, 0.0 };
    point_t const c{ 0.0, 9.0, 0.0 };
    point_t const d{ 0.0, 0.0, 9.0 };
    
    // asserting logical basis for insphere results
    assert(orient<real>(a, b, c, d) == orientation::positive);
    
    point_t const x{ 3.0, 3.0, 3.0 };
    assert(insphere<real>(a, b, c, d, x) == orientation::negative);
    
    point_t const y{ 1000.0, 1000.0, 1000.0 };
    assert(insphere<real>(a, b, c, d, y) == orientation::positive);
    
    point_t const z = b;
    assert(insphere<real>(a, b, c, d, z) == orientation::zero);
  }
  
  // Okay, now for location code testing!
  // For any tetrahedron, we have:
  // 4 vertices
  // 4 faces
  // 6 edges
  // an internal region
  // outside the tetrahedron
  // We must make sure our loc routine can accurately
  // solve these
  {
    point_t const a{ 0.0, 0.0, 0.0 };
    point_t const b{ 9.0, 0.0, 0.0 };
    point_t const c{ 0.0, 9.0, 0.0 };
    point_t const d{ 0.0, 0.0, 9.0 };
    
    assert(orient<real>(a, b, c, d) == orientation::positive);
    
    // We should be able to accurately determine all 6 edge intersections
    {
      point_t const e10{ 4.5, 0.0, 0.0 };
      point_t const e20{ 0.0, 4.5, 0.0 };
      point_t const e30{ 0.0, 0.0, 4.5 };
      point_t const e21{ 4.5, 4.5, 0.0 };
      point_t const e31{ 4.5, 0.0, 4.5 };
      point_t const e23{ 0.0, 4.5, 4.5 };
            
      assert((
        eq<real>(det(matrix<real, 4, 4>{ 1, 0, 0, 0,
                                         1, 0, 9, 0,
                                         1, 0, 0, 9,
                                         1, 4.5, 0, 0 }), 364.5)));
                                         
      assert(orient<real>(d, c, b, e10) == orientation::positive);
      assert(orient<real>(a, c, d, e10) == orientation::positive);
      assert(orient<real>(a, d, b, e10) == orientation::zero);
      assert(orient<real>(a, b, c, e10) == orientation::zero);
      
      assert(loc<real>(a, b, c, d, e10) == 3);
      assert(loc<real>(a, b, c, d, e20) == 5);
      assert(loc<real>(a, b, c, d, e30) == 9);
      assert(loc<real>(a, b, c, d, e21) == 6);
      assert(loc<real>(a, b, c, d, e31) == 10);
      assert(loc<real>(a, b, c, d, e23) == 12);
    }
  }
}

__global__
void tetra_tests_kernel(void)
{
  tetra_tests_impl();
}

auto tetra_tests(void) -> void
{
  std::cout << "Beginning tetra tests!" << std::endl;
  
  tetra_tests_impl();
  
  tetra_tests_kernel<<<1, 256>>>();
  hipDeviceSynchronize();
  
  std::cout << "All tests passed\n" << std::endl;
}
